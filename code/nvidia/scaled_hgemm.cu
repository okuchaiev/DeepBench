#include "hip/hip_runtime.h"
#include "scaled_hgemm.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define min_represent 0.00001526624f

__global__ void createDAScalingMatrices(const int rows, const int cols, const __half *A, __half *DA, __half *invDA) {
	int cur_row_id = blockIdx.x*blockDim.x + threadIdx.x;
	if (cur_row_id<rows){
		//half mx = A[IDX2C(cur_row_id, 0, rows)];
		float mx = fabs(__half2float(A[IDX2C(cur_row_id, 0, rows)]));
		for (int j=1; j<cols; ++j) {
			if (mx< fabs(__half2float(A[IDX2C(cur_row_id, j, rows)])))
				mx = fabs(__half2float(A[IDX2C(cur_row_id, j, rows)]));
		}
		DA[IDX2C(cur_row_id, cur_row_id, rows)] = (mx <= min_represent ? __float2half(1.f) :__float2half(mx));
		invDA[IDX2C(cur_row_id, cur_row_id, rows)] = (mx <= min_represent ? __float2half(1.f) : __float2half(1.f/mx));
	}
};

__global__ void createDBScalingMatrices(const int rows, const int cols, const __half *B, __half *DB, __half *invDB) {
	int cur_col_id = blockIdx.x*blockDim.x + threadIdx.x;
	if (cur_col_id<cols){
		float mx = fabs(__half2float(B[IDX2C(0, cur_col_id, rows)]));
		for (int i=1; i<rows; ++i) {
			if (mx < fabs(__half2float(B[IDX2C(i, cur_col_id, rows)])))
				mx = fabs(__half2float(B[IDX2C(i, cur_col_id, rows)]));
		}
		DB[IDX2C(cur_col_id, cur_col_id, cols)] = (mx <= min_represent ? __float2half(1.f) :__float2half(mx));;
		invDB[IDX2C(cur_col_id, cur_col_id, cols)] = (mx <= min_represent ? __float2half(1.f) : __float2half(1.f/mx));
	}
};

__global__ void zeroInit(int n, __half* data) {
	int ind = blockIdx.x*blockDim.x + threadIdx.x;
	if (ind<n) {
		data[ind] = __float2half(0.f);
	}
}

__global__ void coefConverterFloat2Half(const float *src, __half *out) {
	(*out)=__float2half(*src);
};


__global__ void transposeKernel(int rows, int cols, const __half *src, __half *dst) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<rows) {
		for (int j=0;j<cols;++j) {
			dst[IDX2C(j,i,cols)] = src[IDX2C(i,j,rows)];
		}
	}
}

__half* get_super_slow_transpose(int rows, int cols, const __half *src) {
	__half *fdata;
	hipMalloc(&fdata,sizeof(__half)*rows*cols);
	transposeKernel<<<(rows + 255)/256, 256>>>(rows, cols, src, fdata);
	return fdata;
}

hipblasStatus_t CUBLASWINAPI scaled_Hgemm (hipblasHandle_t handle,
                                                      hipblasOperation_t transa,
                                                      hipblasOperation_t transb,
                                                      int m,
                                                      int n,
                                                      int k,
                                                      const float *alpha,
                                                      const __half *A,
                                                      int lda,
                                                      const __half *B,
                                                      int ldb,
                                                      const float *beta,
                                                      __half *C,
                                                      int ldc, bool raw_hgemm) {


	float *d_alpha, *d_beta;
	hipMalloc(&d_alpha, sizeof(float)); //
	hipMalloc(&d_beta, sizeof(float));  //
	hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

	__half *d_h_alpha, *d_h_beta;
	hipMalloc(&d_h_alpha, sizeof(__half)); //
	hipMalloc(&d_h_beta, sizeof(__half));  //

	coefConverterFloat2Half<<<1,1>>>(d_alpha, d_h_alpha);
	coefConverterFloat2Half<<<1,1>>>(d_beta, d_h_beta);

	hipblasStatus_t status;
	if (raw_hgemm) {
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
		status = hipblasHgemm(handle,
				transa, transb,
				m, n, k,
				d_h_alpha,
				A, lda,
				B, ldb,
				d_h_beta,
				C, ldc
				);
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

	} else { //do outside scaling algorithm
		//bookkeeping for coefficients
		float sp_alpha = 1.f, sp_beta = 0.f;
		float *sp_d_alpha, *sp_d_beta;
		hipMalloc(&sp_d_alpha, sizeof(float)); //
		hipMalloc(&sp_d_beta, sizeof(float));  //
		hipMemcpy(sp_d_alpha, &sp_alpha, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(sp_d_beta, &sp_beta, sizeof(float), hipMemcpyHostToDevice);

		__half *sp_d_h_alpha, *sp_d_h_beta;
		hipMalloc(&sp_d_h_alpha, sizeof(__half));//
		hipMalloc(&sp_d_h_beta, sizeof(__half));//

		coefConverterFloat2Half<<<1,1>>>(sp_d_alpha, sp_d_h_alpha);
		coefConverterFloat2Half<<<1,1>>>(sp_d_beta, sp_d_h_beta);
		//end of coef bookkeeping

		__half *Da, *invDa, *Db, *invDb;
		__half *inptA;
		__half *inptB;

		//Da and invDa
		hipMalloc(&Da, m*m*sizeof(__half));//
		hipMalloc(&invDa, m*m*sizeof(__half));//
		//Db and invDb
		hipMalloc(&Db, n*n*sizeof(__half));//
		hipMalloc(&invDb, n*n*sizeof(__half));//
		zeroInit<<<(m*m+255)/256, 256>>>(m*m, Da);
		zeroInit<<<(m*m+255)/256, 256>>>(m*m, invDa);
		zeroInit<<<(n*n+255)/256, 256>>>(n*n, Db);
		zeroInit<<<(n*n+255)/256, 256>>>(n*n, invDb);
		if (transa==HIPBLAS_OP_T) {
			inptA = get_super_slow_transpose(k, m, A);
			createDAScalingMatrices<<<(m+255)/256,256>>>(m, k, inptA, Da, invDa);
		} else {
			createDAScalingMatrices<<<(m+255)/256,256>>>(m, k, A, Da, invDa);
		}

		if (transb==HIPBLAS_OP_T) {
			inptB = get_super_slow_transpose(n, k, B);
			createDBScalingMatrices<<<(n+255)/256,256>>>(k, n, inptB, Db, invDb);
		} else {
			createDBScalingMatrices<<<(n+255)/256,256>>>(k, n, B, Db, invDb);
		}

		hipDeviceSynchronize();
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

		//Step1&2: Aprime=invDa*A
		__half *Aprime;
		hipMalloc(&Aprime, m*k*sizeof(__half));//
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						m, k, m,
						sp_d_h_alpha,
						invDa, m,
						(transa==HIPBLAS_OP_T ? inptA : A), m,
						sp_d_h_beta,
						Aprime, m
						);
		hipDeviceSynchronize();
		//Steps3&4: Bprime=B*invDb
		__half *Bprime;
		hipMalloc(&Bprime, k*n*sizeof(__half));//
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						k, n, n,
						sp_d_h_alpha,
						(transb==HIPBLAS_OP_T ? inptB : B), k,
						invDb, n,
						sp_d_h_beta,
						Bprime, k
						);
		hipDeviceSynchronize();
		//Step 5
		__half *Cprime;
		hipMalloc(&Cprime, m*n*sizeof(__half));//
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						m, n, k,
						sp_d_h_alpha,
						Aprime, m,
						Bprime, k,
						sp_d_h_beta,
						Cprime, m
						);
		hipDeviceSynchronize();
		//Step 6.1. Cprimeprime = Da*Cprime
		__half *Cprimeprime;
		hipMalloc(&Cprimeprime, m*n*sizeof(__half));//
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						m, n, m,
						sp_d_h_alpha,
						Da, m,
						Cprime, m,
						sp_d_h_beta,
						Cprimeprime, m
						);
		hipDeviceSynchronize();
		//Step 6.2. C = Cprimeprime * Db
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						m, n, n,
						d_h_alpha,
						Cprimeprime, m,
						Db, n,
						d_h_beta,
						C, m
						);
		hipDeviceSynchronize();
		//cleanup
		hipFree(sp_d_alpha);//
		hipFree(sp_d_beta);//
		hipFree(sp_d_h_alpha);//
		hipFree(sp_d_h_beta);//
		hipFree(Da);//
		hipFree(invDa);//
		hipFree(Db);//
		hipFree(invDb);//
		hipFree(Aprime);//
		hipFree(Bprime);//
		hipFree(Cprime);//
		hipFree(Cprimeprime);//
		if (transa==HIPBLAS_OP_T) {
			hipFree(inptA);
		}
		if (transb==HIPBLAS_OP_T) {
			hipFree(inptB);
		}
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

	}//end of outside scaling
	hipFree(d_alpha);//
	hipFree(d_beta);//
	hipFree(d_h_alpha);//
	hipFree(d_h_beta);//
	return status;
}
