#include "hip/hip_runtime.h"
#include "scaled_hgemm.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define min_represent 0.00001526624f

/**
 * Call like this <<<(rows+255)/256, 256>>> if reduce_cols = true
 * <<<(cols+255)/256, 256>>> else
 * inpt still has ld = rows regardless of reduce_cols
 */
__global__ void createScalingDiagonal(const int rows, const int cols, const __half *inpt, __half *res, bool reduce_cols) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (reduce_cols && id < rows) {
		float mx = fabs(__half2float(inpt[IDX2C(id, 0, rows)]));
		for (int j=1; j<cols; ++j) {
			if (mx < fabs(__half2float(inpt[IDX2C(id, j, rows)])))
				mx = fabs(__half2float(inpt[IDX2C(id, j, rows)]));
		}
		res[id] = (mx <= min_represent ? __float2half(1.f) :__float2half(mx));

	} else if (!reduce_cols && id < cols) {
		float mx = fabs(__half2float(inpt[IDX2C(0, id, rows)]));
		for (int i=1; i<rows; ++i) {
			if (mx < fabs(__half2float(inpt[IDX2C(i, id, rows)])))
				mx = fabs(__half2float(inpt[IDX2C(i, id, rows)]));
			}
		res[id] = (mx <= min_represent ? __float2half(1.f) :__float2half(mx));
	}
}

//* Call like this <<<(rows+255)/256, 256>>> u
__global__ void do_scaling(const int rows, const int cols, __half *data, const __half* scales, bool left, bool inv_scale) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	float scale_factor = (!inv_scale ? __half2float(scales[id]) : 1.f/__half2float(scales[id]));
	if (left && id < rows) {
		for (int j=0; j<cols; ++j)
			data[IDX2C(id, j, rows)] = __float2half(__half2float(data[IDX2C(id, j, rows)])* scale_factor);
	} else if (!left && id < cols) {
		for (int i=0; i<rows; ++i)
			data[IDX2C(i, id, rows)] = __float2half(__half2float(data[IDX2C(i, id, rows)])* scale_factor);
	}
}

//Does out=out*beta+arg
__global__ void scale_add(const int rows, const int cols, const __half *arg, __half *out, float beta) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id<rows*cols) {
		out[id] = __float2half((beta != 0.f ? __half2float(out[id])*beta + __half2float(arg[id]) : __half2float(arg[id])));
	}
}


/*
__global__ void createDAScalingMatrices(const int rows, const int cols, const __half *A, __half *DA, __half *invDA) {
	int cur_row_id = blockIdx.x*blockDim.x + threadIdx.x;
	if (cur_row_id<rows){
		//half mx = A[IDX2C(cur_row_id, 0, rows)];
		float mx = fabs(__half2float(A[IDX2C(cur_row_id, 0, rows)]));
		for (int j=1; j<cols; ++j) {
			if (mx< fabs(__half2float(A[IDX2C(cur_row_id, j, rows)])))
				mx = fabs(__half2float(A[IDX2C(cur_row_id, j, rows)]));
		}
		DA[IDX2C(cur_row_id, cur_row_id, rows)] = (mx <= min_represent ? __float2half(1.f) :__float2half(mx));
		invDA[IDX2C(cur_row_id, cur_row_id, rows)] = (mx <= min_represent ? __float2half(1.f) : __float2half(1.f/mx));
	}
};

__global__ void createDBScalingMatrices(const int rows, const int cols, const __half *B, __half *DB, __half *invDB) {
	int cur_col_id = blockIdx.x*blockDim.x + threadIdx.x;
	if (cur_col_id<cols){
		float mx = fabs(__half2float(B[IDX2C(0, cur_col_id, rows)]));
		for (int i=1; i<rows; ++i) {
			if (mx < fabs(__half2float(B[IDX2C(i, cur_col_id, rows)])))
				mx = fabs(__half2float(B[IDX2C(i, cur_col_id, rows)]));
		}
		DB[IDX2C(cur_col_id, cur_col_id, cols)] = (mx <= min_represent ? __float2half(1.f) :__float2half(mx));;
		invDB[IDX2C(cur_col_id, cur_col_id, cols)] = (mx <= min_represent ? __float2half(1.f) : __float2half(1.f/mx));
	}
};

__global__ void zeroInit(int n, __half* data) {
	int ind = blockIdx.x*blockDim.x + threadIdx.x;
	if (ind<n) {
		data[ind] = __float2half(0.f);
	}
}*/

__global__ void coefConverterFloat2Half(const float *src, __half *out) {
	(*out)=__float2half(*src);
};


__global__ void transposeKernel(int rows, int cols, const __half *src, __half *dst) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<rows) {
		for (int j=0;j<cols;++j) {
			dst[IDX2C(j,i,cols)] = src[IDX2C(i,j,rows)];
		}
	}
}

__half* get_super_slow_transpose(int rows, int cols, const __half *src) {
	__half *fdata;
	hipMalloc(&fdata,sizeof(__half)*rows*cols);
	transposeKernel<<<(rows + 255)/256, 256>>>(rows, cols, src, fdata);
	return fdata;
}

hipblasStatus_t CUBLASWINAPI scaled_Hgemm (hipblasHandle_t handle,
                                                      hipblasOperation_t transa,
                                                      hipblasOperation_t transb,
                                                      int m,
                                                      int n,
                                                      int k,
                                                      const float *alpha,
                                                      const __half *A,
                                                      int lda,
                                                      const __half *B,
                                                      int ldb,
                                                      const float *beta,
                                                      __half *C,
                                                      int ldc, bool raw_hgemm) {


	float *d_alpha, *d_beta;
	hipMalloc(&d_alpha, sizeof(float)); //
	hipMalloc(&d_beta, sizeof(float));  //
	hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

	__half *d_h_alpha, *d_h_beta;
	hipMalloc(&d_h_alpha, sizeof(__half)); //
	hipMalloc(&d_h_beta, sizeof(__half));  //

	coefConverterFloat2Half<<<1,1>>>(d_alpha, d_h_alpha);
	coefConverterFloat2Half<<<1,1>>>(d_beta, d_h_beta);

	hipblasStatus_t status;
	if (raw_hgemm) {
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
		status = hipblasHgemm(handle,
				transa, transb,
				m, n, k,
				d_h_alpha,
				A, lda,
				B, ldb,
				d_h_beta,
				C, ldc
				);
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

	} else { //do outside scaling algorithm
		__half *inptA;
		__half *inptB;
		__half *Da, *Db;
		hipMalloc(&Da, sizeof(__half)*m);
		hipMalloc(&Db, sizeof(__half)*n);

		__half *Aprime;
		hipMalloc(&Aprime, m*k*sizeof(__half));
		__half *Bprime;
		hipMalloc(&Bprime, k*n*sizeof(__half));

		//bookkeeping for coefficients
		float sp_alpha = 1.f, sp_beta = 0.f;
		float *sp_d_alpha, *sp_d_beta;
		hipMalloc(&sp_d_alpha, sizeof(float)); //
		hipMalloc(&sp_d_beta, sizeof(float));  //
		hipMemcpy(sp_d_alpha, &sp_alpha, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(sp_d_beta, &sp_beta, sizeof(float), hipMemcpyHostToDevice);

		__half *sp_d_h_alpha, *sp_d_h_beta;
		hipMalloc(&sp_d_h_alpha, sizeof(__half));//
		hipMalloc(&sp_d_h_beta, sizeof(__half));//

		coefConverterFloat2Half<<<1,1>>>(sp_d_alpha, sp_d_h_alpha);
		coefConverterFloat2Half<<<1,1>>>(sp_d_beta, sp_d_h_beta);
		hipDeviceSynchronize();
		//end of coef bookkeeping

		if (transa==HIPBLAS_OP_T) {
			//inptA = get_super_slow_transpose(k, m, A);
			inptA = get_super_slow_transpose(k, m, A);
			hipDeviceSynchronize();
			createScalingDiagonal<<<(m+255)/256, 256>>>(m, k, inptA, Da, true);
			hipDeviceSynchronize();
			hipMemcpy(Aprime, inptA, sizeof(__half)*m*k, hipMemcpyDeviceToDevice);
		} else {
			createScalingDiagonal<<<(m+255)/256, 256>>>(m, k, A, Da, true);
			hipDeviceSynchronize();
			hipMemcpy(Aprime, A, sizeof(__half)*m*k, hipMemcpyDeviceToDevice);
		}

		if (transb==HIPBLAS_OP_T) {
			//inptB = get_super_slow_transpose(n, k, B);
			inptB = get_super_slow_transpose(n, k, B);
			hipDeviceSynchronize();
			createScalingDiagonal<<<(n+255)/256, 256>>>(k, n, inptB, Db, false);
			hipDeviceSynchronize();
			hipMemcpy(Bprime, inptB, sizeof(__half)*n*k, hipMemcpyDeviceToDevice);
		} else {
			createScalingDiagonal<<<(n+255)/256, 256>>>(k, n, B, Db, false);
			hipDeviceSynchronize();
			hipMemcpy(Bprime, B, sizeof(__half)*n*k, hipMemcpyDeviceToDevice);
		}
		hipDeviceSynchronize();

		do_scaling<<<(m+255)/256, 256>>>(m, k, Aprime, Da, true, true);
		hipDeviceSynchronize();
		do_scaling<<<(n+255)/256, 256>>>(k, n, Bprime, Db, false, true);

		__half *Cprime;
		hipMalloc(&Cprime, m*n*sizeof(__half));
		hipDeviceSynchronize();
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						m, n, k,
						sp_d_h_alpha,
						Aprime, m,
						Bprime, k,
						sp_d_h_beta,
						Cprime, m);
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
		hipDeviceSynchronize();
		do_scaling<<<(m+255)/256, 256>>>(m, n, Cprime, Da, true, false);
		hipDeviceSynchronize();
		do_scaling<<<(n+255)/256, 256>>>(m, n, Cprime, Db, false, false);
		hipDeviceSynchronize();
		scale_add<<<(m*n)/256, 256>>>(m, n, Cprime, C, *beta);

		//cleanup
		hipFree(sp_d_alpha);//
		hipFree(sp_d_beta);//
		hipFree(sp_d_h_alpha);//
		hipFree(sp_d_h_beta);//
		hipFree(Da);//
		hipFree(Db);//
		hipFree(Aprime);//
		hipFree(Bprime);//
		hipFree(Cprime);//
		if (transa==HIPBLAS_OP_T) {
			hipFree(inptA);
		}
		if (transb==HIPBLAS_OP_T) {
			hipFree(inptB);
		}
	}//end of outside scaling
	hipFree(d_alpha);//
	hipFree(d_beta);//
	hipFree(d_h_alpha);//
	hipFree(d_h_beta);//
	return status;
}
