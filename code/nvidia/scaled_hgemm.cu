#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <tuple>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include "tensor.h"

hipblasStatus_t CUBLASWINAPI scaled_Hgemm (hipblasHandle_t handle,
                                                      hipblasOperation_t transa,
                                                      hipblasOperation_t transb,
                                                      int m,
                                                      int n,
                                                      int k,
                                                      //const __half *alpha, /* host or device pointer */
                                                      const float *alpha,
                                                      const __half *A,
                                                      int lda,
                                                      const __half *B,
                                                      int ldb,
                                                      //const __half *beta, /* host or device pointer */
                                                      const float *beta,
                                                      __half *C,
                                                      int ldc) {


	return cublasSgemmEx(handle,
						   transa,
	                       transb,
	                       m,
	                       n,
	                       k,
	                       alpha, // host or device pointer
	                       A,
	                       HIP_R_16F,
	                       lda,
	                       B,
	                       HIP_R_16F,
	                       ldb,
	                       beta, // host or device pointer
	                       C,
	                       HIP_R_16F,
	                       ldc);


/*return hipblasHgemm(handle,
            transa,
            transb,
            m,
            n,
            k,
            alpha,
            A, lda,
            B, ldb,
            beta,
            C, ldc);*/
}
