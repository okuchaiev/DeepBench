#include "hip/hip_runtime.h"
#include "scaled_hgemm.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define min_represent 0.0000152663f
#define DivCnst 256



inline void gpuErrchk(hipError_t code, char *label)
{
   if (code != hipSuccess)
   {
	  std::cout<<std::endl<<hipGetErrorString(code)<<"  LABEL: "<<label<<std::endl;
      //exit(code);
   }
}

/**
 * Call like this <<<(rows+255)/256, 256>>> if reduce_cols = true
 * else <<<(cols+255)/256, 256>>>
 * inpt still has ld = rows regardless of reduce_cols
 */

//(k, n, B, Db, Bprime, false, transb==HIPBLAS_OP_T);
__global__ void createScalingDiagonal(const int rows, const int cols, const __half *inpt, __half *res, __half* scaled_inpt, bool reduce_cols, bool transpose_input) {

	int id = blockIdx.x*blockDim.x + threadIdx.x; //row index if reduce_cols = true, else column index

	if (reduce_cols && id < rows) { //id is row index
		float mx = (!transpose_input ? fabs(__half2float(inpt[IDX2C(id, 0, rows)])) : fabs(__half2float(inpt[IDX2C(0, id, cols)])));

		for (int j=1; j<cols; ++j) {
			float element = (!transpose_input ? fabs(__half2float(inpt[IDX2C(id, j, rows)])) : fabs(__half2float(inpt[IDX2C(j, id, cols)])));
			if (mx < element)
				mx = element;
		}
		float scale = (mx <= min_represent ? 1.f : mx);
		res[id] = __float2half(scale);

		for (int j=0; j<cols; ++j) {
			float element = (!transpose_input ? __half2float(inpt[IDX2C(id, j, rows)]) : __half2float(inpt[IDX2C(j, id, cols)]));
			scaled_inpt[IDX2C(id, j, rows)] = __float2half(element/scale);
		}


	} else if (!reduce_cols && id < cols) { //id is column index
		float mx = (!transpose_input ? fabs(__half2float(inpt[IDX2C(0, id, rows)])) : fabs(__half2float(inpt[IDX2C(id, 0, cols)])) );

		for (int i=1; i<rows; ++i) {
			float element = (!transpose_input ? fabs(__half2float(inpt[IDX2C(i, id, rows)])) : fabs(__half2float(inpt[IDX2C(id, i, cols)])));
			if (mx < element)
				mx = element;
			}
		float scale = (mx <= min_represent ? 1.f : mx);
		res[id] = __float2half(scale);

		for (int i=0; i<rows; ++i) {
			float element = (!transpose_input ? __half2float(inpt[IDX2C(i, id, rows)]) : __half2float(inpt[IDX2C(id, i, cols)]));
			scaled_inpt[IDX2C(i, id, rows)] = __float2half(element/scale);
		}
	}
}

//* Call like this <<<(rows+255)/256, 256>>> if left
//  else <<<(cols+255)/256, 256>>>
// Left means scales*data, right means data*scales
__global__ void do_scaling(const int rows, const int cols, __half *data, const __half* scales, bool left, bool inv_scale) {

	int id = blockIdx.x*blockDim.x + threadIdx.x; //row index in data if left, else this is a column index
	float scale_factor = (!inv_scale ? __half2float(scales[id]) : 1.f/__half2float(scales[id]));

	if (left && id < rows) {
		for (int j=0; j<cols; ++j)
			data[IDX2C(id, j, rows)] = __float2half(__half2float(data[IDX2C(id, j, rows)]) * scale_factor);
	} else if (!left && id < cols) {
		for (int i=0; i<rows; ++i)
			data[IDX2C(i, id, rows)] = __float2half(__half2float(data[IDX2C(i, id, rows)]) * scale_factor);
	}
}

//Does out=out*beta+arg
__global__ void scale_add(const int rows, const int cols, const __half *arg, __half *out, float beta) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id<rows*cols) {
		out[id] = __float2half((beta != 0.f ? __half2float(out[id])*beta + __half2float(arg[id]) : __half2float(arg[id])));
	}
}

__global__ void coefConverterFloat2Half(const float *src, __half *out) {
	(*out)=__float2half(*src);
};

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

hipblasStatus_t CUBLASWINAPI scaled_Hgemm (hipblasHandle_t handle,
                                                      hipblasOperation_t transa,
                                                      hipblasOperation_t transb,
                                                      int m,
                                                      int n,
                                                      int k,
                                                      const float *alpha,
                                                      const __half *A,
                                                      int lda,
                                                      const __half *B,
                                                      int ldb,
                                                      const float *beta,
                                                      __half *C,
                                                      int ldc, bool raw_hgemm) {

	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	float *d_alpha, *d_beta;
	gpuErrchk(hipMalloc((void **)&d_alpha, sizeof(float)), "1");
	gpuErrchk(hipMalloc((void **)&d_beta, sizeof(float)), "2");
	hipMemcpy(d_alpha, alpha, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice);

	__half *d_h_alpha, *d_h_beta;
	gpuErrchk(hipMalloc((void **)&d_h_alpha, sizeof(__half)), "3");
	gpuErrchk(hipMalloc((void **)&d_h_beta, sizeof(__half)), "4");

	coefConverterFloat2Half<<<1,1>>>(d_alpha, d_h_alpha);
	gpuErrchk( hipPeekAtLastError() , "5");
	gpuErrchk( hipDeviceSynchronize(), "6" );

	coefConverterFloat2Half<<<1,1>>>(d_beta, d_h_beta);
	gpuErrchk( hipPeekAtLastError(), "7" );
	gpuErrchk( hipDeviceSynchronize(), "8" );

	hipblasStatus_t status;
	if (raw_hgemm) {
		status = hipblasHgemm(handle,
				transa, transb,
				m, n, k,
				d_h_alpha,
				A, lda,
				B, ldb,
				d_h_beta,
				C, ldc);
		//hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

	} else { //do outside scaling algorithm
		__half *Da, *Db, *Aprime, *Bprime, *Cprime;
		gpuErrchk(hipMalloc((void **)&Da, sizeof(__half)*m), "9");//
		gpuErrchk(hipMalloc((void **)&Db, sizeof(__half)*n), "10");//
		gpuErrchk(hipMalloc((void **)&Aprime, m*k*sizeof(__half)), "11");//
		gpuErrchk(hipMalloc((void **)&Bprime, k*n*sizeof(__half)), "12");//
		gpuErrchk(hipMalloc((void **)&Cprime, m*n*sizeof(__half)), "13");//

		//bookkeeping for coefficients
		float sp_beta = 0.f; //host
		float  *sp_d_beta; //device
		gpuErrchk(hipMalloc((void **)&sp_d_beta, sizeof(float)), "14");
		gpuErrchk(hipMemcpy(sp_d_beta, &sp_beta, sizeof(float), hipMemcpyHostToDevice), "15");
		__half *sp_d_h_beta;
		gpuErrchk(hipMalloc((void **)&sp_d_h_beta, sizeof(__half)), "16");
		coefConverterFloat2Half<<<1,1>>>(sp_d_beta, sp_d_h_beta);
		gpuErrchk( hipPeekAtLastError(), "17" );
		gpuErrchk( hipDeviceSynchronize(), "18" );

		//Step 1&2
		createScalingDiagonal<<<(m+DivCnst-1)/DivCnst, DivCnst>>>(m, k, A, Da, Aprime, true, transa==HIPBLAS_OP_T);
		gpuErrchk( hipPeekAtLastError(), "19" );
		gpuErrchk( hipDeviceSynchronize(), "20" );
		//Step 3&4
		createScalingDiagonal<<<(n+DivCnst-1)/DivCnst, DivCnst>>>(k, n, B, Db, Bprime, false, transb==HIPBLAS_OP_T);
		gpuErrchk( hipPeekAtLastError(), "21" );
		gpuErrchk( hipDeviceSynchronize(), "22" );

		//Step 5
		status = hipblasHgemm(handle,
						HIPBLAS_OP_N, HIPBLAS_OP_N,//transa, transb,
						m, n, k,
						d_h_alpha,
						Aprime, lda,
						//Aprime, m,
						//Bprime, k,
						Bprime, ldb,
						sp_d_h_beta,
						//Cprime, m);
						Cprime, ldc);

		//hipFree(sp_d_alpha);//
		hipFree(sp_d_beta);//
		//hipFree(sp_d_h_alpha);//
		hipFree(sp_d_h_beta);//
		hipFree(Aprime);//
		hipFree(Bprime);//
		do_scaling<<<(m+DivCnst-1)/DivCnst, DivCnst>>>(m, n, Cprime, Da, true, false);
		hipDeviceSynchronize();
		do_scaling<<<(n+DivCnst-1)/DivCnst, DivCnst>>>(m, n, Cprime, Db, false, false);
		hipDeviceSynchronize();
		scale_add<<<(m*n+DivCnst-1)/DivCnst, DivCnst>>>(m, n, Cprime, C, *beta);
		//cleanup
		hipFree(Da);//
		hipFree(Db);//
		hipFree(Cprime);//
	}//end of outside scaling
	hipFree(d_alpha);//
	hipFree(d_beta);//
	hipFree(d_h_alpha);//
	hipFree(d_h_beta);//
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
	//checkCudaErrors();
	if (status!=HIPBLAS_STATUS_SUCCESS) {
		std::cout << std::endl << std::endl;
		std::cout << _cudaGetErrorEnum(status) << std::endl;

	}
	return status;
	//return HIPBLAS_STATUS_SUCCESS;
}
